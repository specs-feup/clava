
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 1 << 5;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    // initialize data at host side

    for (int i = 0; i < nElem; i++)
    {
        h_A[i] = h_B[i] = i;
    }

    memset(gpuRef,  0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    dim3 block (nElem);
    dim3 grid  (1);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(gpuRef);

    hipDeviceReset();
    return(0);
}