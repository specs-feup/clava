
#include <hip/hip_runtime.h>
#include <stdio.h>
#define TILE_DIM 16

__global__ void multiMatrix (int *a, int *b, int *c, int N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;                
	int idy = threadIdx.y + blockDim.y * blockIdx.y;                
    int pos = idx + idy * N;                                        
	int temp_result = 0;  
	int posa, posb;                                          

	__shared__ int s_a[TILE_DIM][TILE_DIM];
	__shared__ int s_b[TILE_DIM][TILE_DIM];

	for (int tile_idx = 0; tile_idx < gridDim.x; tile_idx++) 
	{
        posa = idy * N + (tile_idx * TILE_DIM + threadIdx.x);
        posb = (tile_idx * TILE_DIM + threadIdx.y) * N + idx;
		
		if (posa < N*N) {
			s_a[threadIdx.y][threadIdx.x] = a[posa];
        } else {
            s_a[threadIdx.y][threadIdx.x] = 0;
        }
        if (posb < N*N) {
            s_b[threadIdx.y][threadIdx.x] = b[posb];
        } else {
            s_b[threadIdx.y][threadIdx.x] = 0;
        }
                                                                    
        __syncthreads();
		if (idx < N && idy < N) {
			for (int i=0; i < TILE_DIM; i++) {
                temp_result += s_a[threadIdx.y][i] * 
					s_b[i][threadIdx.x]; 
			}
		}
            
        __syncthreads();  
    }
    __syncthreads();   
    
    if(idx < N && idy < N)  {
		c[pos] = temp_result;
	} 
} 

int main (int argc, char* argv[]){
	int N = 4; 
	size_t size = N*N*sizeof(int);
	int num_thread, num_block;

	int *h_a, *h_b, *h_c;
	h_a = (int*)malloc(size);
	h_b = (int*)malloc(size);
	h_c = (int*)malloc(size);

	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

    int i = 0, j = 0;
	for (i = 0; i < N*N; i++){
		h_a[i] = h_b[i] = i;
	}
	
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);
	hipMemset(d_c,0,size);

	num_block = ceil((float)N/TILE_DIM);
    num_thread = N < TILE_DIM ? N : TILE_DIM;

	printf("Blocks: %d    Threads: %d  \n", num_block, num_thread);

	dim3 gridsize(num_block,num_block,1);
	dim3 blocksize(num_thread,num_thread,1);
  
    multiMatrix<<<gridsize,blocksize>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);


	hipFree(d_a);
	hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
}