
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = id % 10;
    //a[idx] += 10;
    atomicAdd(&a[idx], 10);
}

int main(){
    int *a;
    size_t size = 10 * sizeof(int);

    hipMallocManaged(&a, size);

    add<<<10,1024>>>(a);
    hipDeviceSynchronize();

    for (int i = 0; i < 10; i++)
        printf("%d ", a[i]);

}