
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NSTREAM 4
#define BDIM 128

void printArray(float *a, int size){
    for (int i = 0; i < size; i++){
        if(i % 128 == 0)
            printf("\n");
        printf("%.0f ", a[i]);
    }
    printf("\n\n");
}


__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        if( idx == N-1)
            C[idx] = A[idx+2] + B[idx+2];
        else
            C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char **argv)
{
    printf("> %s Starting...\n", argv[0]);

    // set up data size of vectors
    int nElem = 1 << 9;
    printf("> vector size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float *h_A, *h_B, *gpuRef;
    hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault);
    hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault);
    hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault);

    // initialize data at host side
    for (int i = 0; i < nElem; i++)
    {
        h_A[i] = h_B[i] = i;
    }
    memset(gpuRef,  0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // invoke kernel at host side
    dim3 block (BDIM);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x,
            block.y);

    // sequential operation
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    printf("\n");
    printArray(gpuRef, nElem);

    // grid parallel operation
    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
    {
        hipStreamCreate(&stream[i]);
    }


    // initiate all work on the device asynchronously in depth-first order
    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * iElem;
        hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]);
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset],
                &d_C[ioffset], iElem);
        hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes,
                              hipMemcpyDeviceToHost, stream[i]);
    }

    // check kernel error
    hipGetLastError();

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(gpuRef);


    // destroy streams
    for (int i = 0; i < NSTREAM; ++i)
    {
        hipStreamDestroy(stream[i]);
    }

    hipDeviceReset();
    return(0);
}