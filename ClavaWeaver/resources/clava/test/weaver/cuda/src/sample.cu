
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sample (int *d_a, int N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;                
	int idy = threadIdx.y + blockDim.y * blockIdx.y;                
    int pos = idx + idy * N;                                                                              

    if(pos % 2 == 0)
        d_a[pos] = 1;
} 

int main (int argc, char* argv[]) {
	int N = 8; 
	size_t size = N*N*sizeof(int);

	int *h_a;
	h_a = (int*)malloc(size);

	int *d_a;
	hipMalloc(&d_a, size);

    int i = 0;
	for (i = 0; i < N*N; i++)
		h_a[i] = 0;
	
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
	sample<<<4,4>>>(d_a, N);
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

	for(i = 0; i < N*N; i++)
		printf("%d ", h_a[i]);

	hipFree(d_a);
    free(h_a);
}